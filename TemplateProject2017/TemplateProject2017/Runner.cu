#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <chrono>
#include <random>

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

// OpenGL Graphics includes
#include <glew.h>
#include <freeglut.h>
#include <cudaDefs.h>
#include <imageManager.h>

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <math_functions.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <timer.h>               // timing functions

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check
#include <helper_gl.h>      // helper functions for CUDA/GL interop

#include "imageKernels.cuh"

#define BLOCK_DIM 8

hipError_t error = hipSuccess;
hipDeviceProp_t deviceProp = hipDeviceProp_t();

//CUDA variables
unsigned int imageWidth;
unsigned int imageHeight;
unsigned int imageBPP;		//Bits Per Pixel = 8, 16, 24, or 32 bit
unsigned int imagePitch;
hipGraphicsResource_t cudaPBOResource;
hipGraphicsResource_t cudaTexResource;
texture<uchar4, 2, hipReadModeElementType> cudaTexRef;
hipChannelFormatDesc cudaTexChannelDesc;
KernelSetting ks;

//OpenGL
unsigned int pboID;
unsigned int textureID;

unsigned int viewportWidth = 1024;
unsigned int viewportHeight = 1024;

//Application
constexpr unsigned int NUM_OF_RAIN_DROPS = 1 << 20;

bool isGradientCalculated = false;

unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
std::mt19937_64 generator(seed);

struct RainDrop {
private:

public:
	RainDrop()
		: gridPosition(int2()), exactPosition(double2()), directionForce(double2()), isDrop(false) {}

	void setPosition(const int posX, const int posY) {
		this->gridPosition = make_int2(posX, posY);
		this->exactPosition = make_double2(posX, posY);
	}
	void setDrop(const bool isDrop) {
		this->isDrop = isDrop;
	}

	bool isDropOnPosition() { return this->isDrop; }

	int2 gridPosition;
	double2 exactPosition;
	double2 directionForce;
	bool isDrop;
};

struct RainBuffer {
	RainBuffer()
		: actualDropState(0), promiseDropState(0), directionForce(int2()) {}

	void addDropsToState(const unsigned int drop) {
		actualDropState += drop;
		promiseDropState += drop;
	}

	int2 directionForce;

	float actualDropState;
	float promiseDropState;
};

RainDrop* hRainDropDevPtr = nullptr;
RainDrop* dRainDropDevPtr = nullptr;

RainBuffer* hRainBufferDevPtr = nullptr;
RainBuffer* dRainBufferDevPtr = nullptr;

size_t pitch;

void cudaWorker();
void loadTexture(const char* imageFileName);
void preparePBO();
void my_display();
void my_resize(GLsizei w, GLsizei h);
void my_idle();
void initGL(int argc, char **argv);
void releaseOpenGL();
void initCUDAtex();
void releaseCUDA();
void releaseApplication();
void releaseResources();
void assignRainBuffer();
void generateRandomDropsPosition(RainBuffer* rainBuffer, const unsigned int numOfDrops, const unsigned int width, const unsigned int height, const unsigned int depth);

__global__ void calculateGradients(const unsigned int pboWidth, const unsigned int pboHeight, unsigned char *pbo, RainBuffer* rainBuffer, const unsigned int pitch) {
	const unsigned int tx = blockDim.x * blockIdx.x + threadIdx.x;
	const unsigned int ty = blockDim.y * blockIdx.y + threadIdx.y;

	if(tx >= pboWidth || ty >= pboHeight) { return; }

	uchar4 tex = tex2D(cudaTexRef, tx, ty);

	const unsigned int rainBufferPitch = pitch / sizeof(RainBuffer);
	const unsigned int rainBufferOffset = ty * rainBufferPitch + tx;

	const unsigned int checkOffset = 1;

	const float lt = tex2D(cudaTexRef, tx - (1 * checkOffset), ty - (1 * checkOffset)).x;
	const float lc = tex2D(cudaTexRef, tx - (1 * checkOffset), ty + (0 * checkOffset)).x;
	const float lb = tex2D(cudaTexRef, tx - (1 * checkOffset), ty + (1 * checkOffset)).x;
	const float ct = tex2D(cudaTexRef, tx + (0 * checkOffset), ty - (1 * checkOffset)).x;
	const float cb = tex2D(cudaTexRef, tx + (0 * checkOffset), ty + (1 * checkOffset)).x;
	const float rt = tex2D(cudaTexRef, tx + (1 * checkOffset), ty - (1 * checkOffset)).x;
	const float rc = tex2D(cudaTexRef, tx + (1 * checkOffset), ty + (0 * checkOffset)).x;
	const float rb = tex2D(cudaTexRef, tx + (1 * checkOffset), ty + (1 * checkOffset)).x;

	int direction = 0;

	if(lt < tex.x) { direction = 1; tex.x = lt; }
	if(lc < tex.x) { direction = 2; tex.x = lc; }
	if(lb < tex.x) { direction = 3; tex.x = lb; }
	if(ct < tex.x) { direction = 4; tex.x = ct; }
	if(cb < tex.x) { direction = 5; tex.x = cb; }
	if(rt < tex.x) { direction = 6; tex.x = rt; }
	if(rc < tex.x) { direction = 7; tex.x = rc; }
	if(rb < tex.x) { direction = 8; tex.x = rb; }

	if(direction == 1) { rainBuffer[rainBufferOffset].directionForce = make_int2(-1, -1); }
	if(direction == 2) { rainBuffer[rainBufferOffset].directionForce = make_int2(-1,  0); }
	if(direction == 3) { rainBuffer[rainBufferOffset].directionForce = make_int2(-1,  1); }
	if(direction == 4) { rainBuffer[rainBufferOffset].directionForce = make_int2( 0, -1); }
	if(direction == 5) { rainBuffer[rainBufferOffset].directionForce = make_int2( 0,  1); }
	if(direction == 6) { rainBuffer[rainBufferOffset].directionForce = make_int2( 1, -1); }
	if(direction == 7) { rainBuffer[rainBufferOffset].directionForce = make_int2( 1,  0); }
	if(direction == 8) { rainBuffer[rainBufferOffset].directionForce = make_int2( 1,  1); }
}

__global__ void moveRainDrops(const unsigned int pboWidth, const unsigned int pboHeight, unsigned char *pbo, RainBuffer* rainBuffer, const unsigned int pitch) {
	const unsigned int tx = blockDim.x * blockIdx.x + threadIdx.x;
	const unsigned int ty = blockDim.y * blockIdx.y + threadIdx.y;

	if(tx >= pboWidth || ty >= pboHeight) { return; }

	const unsigned int rainBufferPitch = pitch / sizeof(RainBuffer);
	const unsigned int rainBufferOffset = ty * rainBufferPitch + tx;

	int2 directionForce = rainBuffer[rainBufferOffset].directionForce;

	const unsigned int flowTo = (ty + directionForce.y) * rainBufferPitch + (tx + directionForce.x);

	if(flowTo > 0 && flowTo < pboWidth * pboHeight) { 
		rainBuffer[flowTo].promiseDropState += rainBuffer[rainBufferOffset].actualDropState * 0.025;

		rainBuffer[rainBufferOffset].promiseDropState -= rainBuffer[rainBufferOffset].actualDropState * 0.025;

		rainBuffer[rainBufferOffset].promiseDropState = max(rainBuffer[rainBufferOffset].promiseDropState, 0.0);
	}
}

__global__ void vizualizeRainDrops(const unsigned int pboWidth, const unsigned int pboHeight, unsigned char *pbo, RainBuffer* rainBuffer, const unsigned int pitch) {
	const unsigned int tx = blockDim.x * blockIdx.x + threadIdx.x;
	const unsigned int ty = blockDim.y * blockIdx.y + threadIdx.y;

	if(tx >= pboWidth || ty >= pboHeight) { return; }

	uchar4 tex = tex2D(cudaTexRef, tx, ty);

	const unsigned int texOffset = (ty * pboWidth + tx) * 4;
	//pbo[texOffset + 0] = tex.x; pbo[texOffset + 1] = tex.y; pbo[texOffset + 2] = tex.z; pbo[texOffset + 3] = tex.w;
	pbo[texOffset + 0] = 0; pbo[texOffset + 1] = 0; pbo[texOffset + 2] = 0; pbo[texOffset + 3] = 0;

	const unsigned int rainBufferPitch = pitch / sizeof(RainBuffer);
	const unsigned int rainBufferOffset = ty * rainBufferPitch + tx;

	rainBuffer[rainBufferOffset].actualDropState = rainBuffer[rainBufferOffset].promiseDropState;

	//pbo[texOffset + 2] = max((int)rainBuffer[rainBufferOffset].actualDropState, pbo[texOffset + 2]);
	if(rainBuffer[rainBufferOffset].actualDropState > 0) { pbo[texOffset + 2] = rainBuffer[rainBufferOffset].actualDropState; }
}

int main(int argc, char *argv[])
{
	initializeCUDA(deviceProp);

	initGL(argc, argv);

	loadTexture("heightmap.png");
	//loadTexture("gradient.gif");

	preparePBO();

	initCUDAtex();

	hRainBufferDevPtr = new RainBuffer[imageWidth * imageHeight];

	generateRandomDropsPosition(hRainBufferDevPtr, NUM_OF_RAIN_DROPS, imageWidth, imageHeight, 1);

	assignRainBuffer();

	//start rendering mainloop
	glutMainLoop();
	atexit(releaseResources);
}

void assignRainBuffer() {
	checkCudaErrors(hipMallocPitch((void**)&dRainBufferDevPtr, &pitch, imageHeight * sizeof(RainBuffer), imageWidth));
	checkCudaErrors(hipMemcpy2D(dRainBufferDevPtr, pitch, hRainBufferDevPtr, imageHeight * sizeof(RainBuffer), imageHeight * sizeof(RainBuffer), imageWidth, hipMemcpyHostToDevice));
}

void generateRandomDropsPosition(RainBuffer* rainBuffer, const unsigned int numOfDrops, const unsigned int width, const unsigned int height, const unsigned int depth) {
	std::uniform_int_distribution<int> dis_x(0, width);
	std::uniform_int_distribution<int> dis_y(0, height);

	for(unsigned int i = 0; i < numOfDrops; i++) {
		int index = (dis_y(generator) * width) + dis_x(generator);

		if(index < width * height)
			rainBuffer[index].addDropsToState(30);
	}
}

void cudaWorker()
{
	hipArray* array;

	//TODO 3: Map cudaTexResource
	hipGraphicsMapResources(1, &cudaTexResource, 0);

	//TODO 4: Get Mapped Array of cudaTexResource
	hipGraphicsSubResourceGetMappedArray(&array, cudaTexResource, 0, 0);

	//TODO 5: Get cudaTexChannelDesc from previously obtained array
	hipGetChannelDesc(&cudaTexChannelDesc, array);

	//TODO 6: Binf cudaTexRef to array
	hipBindTextureToArray(&cudaTexRef, array, &cudaTexChannelDesc);
	checkError();

	unsigned char *pboData;
	size_t pboSize;
	//TODO 7: Map cudaPBOResource
	hipGraphicsMapResources(1, &cudaPBOResource, 0);

	//TODO 7: Map Mapped pointer to cudaPBOResource data
	hipGraphicsResourceGetMappedPointer((void**)&pboData, &pboSize, cudaPBOResource);
	checkError();

	//TODO 8: Set KernelSetting variable ks (dimBlock, dimGrid, etc.) such that block will have BLOCK_DIM x BLOCK_DIM threads
	ks.blockSize = BLOCK_DIM * BLOCK_DIM;
	ks.dimBlock = dim3(BLOCK_DIM, BLOCK_DIM, 1);
	ks.dimGrid = dim3((imageWidth + BLOCK_DIM - 1) / BLOCK_DIM, (imageHeight + BLOCK_DIM - 1) / BLOCK_DIM, 1);

	//Calling kernels
	if(!isGradientCalculated) {
		calculateGradients << <ks.dimGrid, ks.dimBlock >> > (imageWidth, imageHeight, pboData, dRainBufferDevPtr, pitch);

		isGradientCalculated = true;
	}

	moveRainDrops << <ks.dimGrid, ks.dimBlock >> > (imageWidth, imageHeight, pboData, dRainBufferDevPtr, pitch);
	vizualizeRainDrops << <ks.dimGrid, ks.dimBlock >> > (imageWidth, imageHeight, pboData, dRainBufferDevPtr, pitch);

	//Following code release mapped resources, unbinds texture and ensures that PBO data will be coppied into OpenGL texture. Do not modify following code!
	hipUnbindTexture(&cudaTexRef);
	hipGraphicsUnmapResources(1, &cudaPBOResource, 0);
	hipGraphicsUnmapResources(1, &cudaTexResource, 0);

	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pboID);
	glBindTexture(GL_TEXTURE_2D, textureID);
	glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, imageWidth, imageHeight, GL_RGBA, GL_UNSIGNED_BYTE, NULL);   //Source parameter is NULL, Data is coming from a PBO, not host memory
}

#pragma region OpenGL Routines - DO NOT MODIFY THIS SECTION !!!

void loadTexture(const char* imageFileName)
{
	FreeImage_Initialise();
	FIBITMAP *tmp = ImageManager::GenericLoader(imageFileName, 0);

	imageWidth = FreeImage_GetWidth(tmp);
	imageHeight = FreeImage_GetHeight(tmp);
	imageBPP = FreeImage_GetBPP(tmp);
	imagePitch = FreeImage_GetPitch(tmp);

	tmp = FreeImage_ConvertTo32Bits(tmp);

	//OpenGL Texture
	glEnable(GL_TEXTURE_2D);
	glGenTextures(1, &textureID);
	glBindTexture(GL_TEXTURE_2D, textureID);

	//WARNING: Just some of inner format are supported by CUDA!!!
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, imageWidth, imageHeight, 0, GL_BGRA, GL_UNSIGNED_BYTE, FreeImage_GetBits(tmp));
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);

	FreeImage_Unload(tmp);
}

void preparePBO()
{
	glGenBuffers(1, &pboID);
	glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pboID);												// Make this the current UNPACK buffer (OpenGL is state-based)
	glBufferData(GL_PIXEL_UNPACK_BUFFER, imageWidth * imageHeight * 4, NULL, GL_DYNAMIC_COPY);	// Allocate data for the buffer. 4-channel 8-bit image
}

void my_display()
{
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glEnable(GL_TEXTURE_2D);
	glBindTexture(GL_TEXTURE_2D, textureID);

	//I know this is a very old OpenGL, but we want to practice CUDA :-)
		//Now it will be a wasted time to learn you current features of OpenGL. Sorry for that however, you can visit my second seminar dealing with Computer Graphics (CG2).
	glBegin(GL_QUADS);

	glTexCoord2d(0, 0);		glVertex2d(0, 0);
	glTexCoord2d(1, 0);		glVertex2d(viewportWidth, 0);
	glTexCoord2d(1, 1);		glVertex2d(viewportWidth, viewportHeight);
	glTexCoord2d(0, 1);		glVertex2d(0, viewportHeight);

	glEnd();

	glDisable(GL_TEXTURE_2D);

	glFlush();
	glutSwapBuffers();
}

void my_resize(GLsizei w, GLsizei h)
{
	viewportWidth = w;
	viewportHeight = h;

	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glViewport(0, 0, viewportWidth, viewportHeight);

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
	gluOrtho2D(0, viewportWidth, 0, viewportHeight);

	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();
	glutPostRedisplay();
}

void my_idle()
{
	cudaWorker();
	glutPostRedisplay();
}

void initGL(int argc, char **argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
	glutInitWindowSize(viewportWidth, viewportHeight);
	glutInitWindowPosition(0, 0);
	glutCreateWindow(":-)");

	glutDisplayFunc(my_display);
	glutReshapeFunc(my_resize);
	glutIdleFunc(my_idle);
	glutSetCursor(GLUT_CURSOR_CROSSHAIR);

	// initialize necessary OpenGL extensions
	glewInit();

	glClearColor(0.0, 0.0, 0.0, 1.0);
	glShadeModel(GL_SMOOTH);
	glViewport(0, 0, viewportWidth, viewportHeight);

	glFlush();
}

void releaseOpenGL()
{
	if(textureID > 0)
		glDeleteTextures(1, &textureID);
	if(pboID > 0)
		glDeleteBuffers(1, &pboID);
}

#pragma endregion

#pragma region CUDA Routines

void initCUDAtex()
{
	cudaGLSetGLDevice(0);
	checkError();

	//CUDA Texture settings
	cudaTexRef.normalized = false;						//Otherwise TRUE to access with normalized texture coordinates
	cudaTexRef.filterMode = hipFilterModePoint;		//Otherwise texRef.filterMode = hipFilterModeLinear; for Linear interpolation of texels
	cudaTexRef.addressMode[0] = hipAddressModeClamp;	//No repeat texture pattern
	cudaTexRef.addressMode[1] = hipAddressModeClamp;	//No repeat texture pattern

	//TODO 1: Register OpenGL texture to CUDA resource
	hipGraphicsGLRegisterImage(&cudaTexResource, textureID, GL_TEXTURE_2D, hipGraphicsRegisterFlagsReadOnly);
	checkError();

	//TODO 2: Register PBO to CUDA resource
	hipGraphicsGLRegisterBuffer(&cudaPBOResource, pboID, hipGraphicsRegisterFlagsWriteDiscard);
	checkError();
}

void releaseCUDA()
{
	hipGraphicsUnregisterResource(cudaPBOResource);
	hipGraphicsUnregisterResource(cudaTexResource);
}

#pragma endregion

void releaseApplication() {
	delete[] hRainBufferDevPtr;
	delete[] dRainBufferDevPtr;
}

void releaseResources()
{
	releaseCUDA();
	releaseOpenGL();
	releaseApplication();
}
